#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <iostream>
#include <fstream>
#include <cmath>
#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <thrust/functional.h>

#include "index.h"

double3 getInfModel()
{
    double3 m;
    m.z = std::numeric_limits<double>::infinity();
    return m;
}

struct dist_functor {
    double slope = 0;
    double intercept = 0;
    dist_functor(const double _slope, const double _intercept)
    {
        slope = _slope;
        intercept = _intercept;
    }

    __host__ __device__
    double operator()(const double2 &p) const
    {
        double dist = std::abs(slope * p.x - p.y + intercept) / std::sqrt(slope * slope + 1);
        return dist * dist;
    }
};

struct min_model
{
    __host__ __device__
    double3 operator()(const double3 &a, const double3 &b) const
    {
        return a.z < b.z ? a : b;
    }
};

struct LinearModel
{
    double slope;
    double intercept;
    double error;
};

struct LinearModel minModel(struct LinearModel a, struct LinearModel b)
{
    return a.error < b.error ? a : b;
}

double distance(struct LinearModel model, double2 p)
{
    return std::abs(model.slope * p.x - p.y + model.intercept) / std::sqrt(model.slope * model.slope + 1);
}

template<class T>
void shuffle(T *arr, int size)
{
    // Fisher Yates Shuffle
    // https://en.wikipedia.org/wiki/Fisher%E2%80%93Yates_shuffle
    for (int i = size - 1; i > 0; i--) 
    { 
        int j = rand() % (i + 1); 
        T tmp = arr[i];
        arr[i] = arr[j];
        arr[j] = tmp;
    }
}

int * buildIndices(int dataSize, int maxIter)
{
    int *indices = (int*)malloc(maxIter * dataSize * sizeof(int));
    for (int iter = 0; iter < maxIter; iter++)
    {
        for (int idx = 0; idx < dataSize; idx++)
        {
            indices[iter * dataSize + idx] = idx;
        }
        shuffle(&indices[iter * dataSize], dataSize);
    }
    return indices;
}

__device__ void triangToTuple(int t, int &x, int &y)
{
    y = (int)((1 + (int)sqrtf(1 + 8 * t)) / 2);
    x = t - (y * (y - 1)) / 2;
}

__global__ void buildModel(double2 *data, int numCombinations, struct LinearModel *models)
{
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    int i, k;
    triangToTuple(t, i, k);

    double2 pI = data[i];
    double2 pK = data[k];

    struct LinearModel m;
    m.slope = (pK.y - pI.y) / (pK.x - pI.x);
    m.intercept = pI.y - m.slope * pI.x;

    models[t] = m;
}

int checkGood(struct LinearModel m, double2 *data, int dataSize, int trainSize, double thresh, int *good)
{
    // reduce
    int goodSize = 0;
    for (int d = trainSize; d < dataSize; d++)
    {
        double2 p = data[d];
        double dist = distance(m, p);
        if (dist < thresh)
        {
            good[goodSize] = d;
            goodSize++;
        }
    }

    return goodSize;
}

struct LinearModel findBest(int trainSize, struct LinearModel *candidates, int candidatesSize, double2 *data)
{
    double3 cs[candidatesSize];

    // reduce
    for (int t = 0; t < candidatesSize; t++)
    {
        struct LinearModel m = candidates[t];
        struct dist_functor dist_op = dist_functor(m.slope, m.intercept);
        thrust::plus<double> add_op;
        double sum = thrust::transform_reduce(data, data + trainSize, dist_op, 0.0, add_op);
        // mean squarred error
        cs[t].x = m.slope;
        cs[t].y = m.intercept;
        cs[t].z = sum / trainSize;
    }

    struct min_model min_op;
    double3 bestM = thrust::reduce(cs, cs + candidatesSize, getInfModel(), min_op);
    struct LinearModel m;
    m.slope = bestM.x;
    m.intercept = bestM.y;
    m.error = bestM.z;
    return m;
}

struct LinearModel singleIter(int iter, double2 *rawData, int dataSize, double thresh, int trainSize, int wellCount)
{
    // create a copy of the data (maybe not needed in the future)
    double2 data[dataSize];
    for (int i = 0; i < dataSize; i++)
    {
        data[i] = rawData[i];
    }
    shuffle(data, dataSize);
    double2 *d_data;
    hipMalloc(&d_data, dataSize * sizeof(double2));
    hipMemcpy(d_data, data, dataSize * sizeof(double2), hipMemcpyHostToDevice);

    // the number of unique combinations of all data points
    int numCombinations = triangMax(trainSize);

    int threadsPerBlock = 32;
    int numBlocks = numCombinations / threadsPerBlock;
    // ignore some combinations OR fill them up with duplicates
    numCombinations = numBlocks * threadsPerBlock;

    // produce every possible model
    int candidateSize = numCombinations * sizeof(struct LinearModel);
    struct LinearModel *d_candidateModels;
    hipMalloc(&d_candidateModels, candidateSize);
    
    buildModel<<<numBlocks, threadsPerBlock>>>(d_data, numCombinations, d_candidateModels);
    struct LinearModel *candidateModels = (struct LinearModel*)malloc(candidateSize);
    hipMemcpy(candidateModels, d_candidateModels, candidateSize, hipMemcpyDeviceToHost);
    hipFree(d_candidateModels);

    struct LinearModel bestModel = findBest(trainSize, candidateModels, numCombinations, data);

    free(candidateModels);

    // evaluate the models
    int numGood;
    int tmpInlierIndices[dataSize - trainSize];
    numGood = checkGood(bestModel, data, dataSize, trainSize, thresh, tmpInlierIndices);
    if (numGood < wellCount)
    {
        struct LinearModel fail;
        fail.slope = 0;
        fail.intercept = 0;
        fail.error = std::numeric_limits<double>::infinity();
        return fail;
    }
    else
    {
        // reorder (put the new inliers first)
        for (int g = 0; g < numGood; g++)
        {
            double2 tmp = data[trainSize + g];
            data[trainSize + g] = data[tmpInlierIndices[g]];
            data[tmpInlierIndices[g]] = tmp;
        }
    }

    int numTrainAndGood = trainSize + numGood;
    int numInlierComb = triangMax(numTrainAndGood);

    // fit again using the new inlier indices
    numBlocks = numInlierComb / threadsPerBlock;
    numInlierComb = numBlocks * threadsPerBlock;
    struct LinearModel *inlierModels = (struct LinearModel*)malloc(numInlierComb * sizeof(struct LinearModel));
    struct LinearModel *d_inlierModels;
    hipMalloc(&d_inlierModels, numInlierComb * sizeof(inlierModels[0]));

    buildModel<<<numBlocks, threadsPerBlock>>>(d_data, numInlierComb, d_inlierModels);

    hipMemcpy(inlierModels, d_inlierModels, numInlierComb * sizeof(inlierModels[0]), hipMemcpyDeviceToHost);
    hipFree(d_inlierModels);

    bestModel = findBest(numTrainAndGood, inlierModels, numInlierComb, data);

    free(inlierModels);

    hipFree(d_data);

    return bestModel;
}

struct LinearModel ransac(double2 *data, int dataSize, int maxIter, double thresh, int trainSize, int wellCount)
{
    struct LinearModel best;
    best.error = std::numeric_limits<double>::infinity();
    for (int iter = 0; iter < maxIter; iter++)
    {
        struct LinearModel m = singleIter(iter, data, dataSize, thresh, trainSize, wellCount);
        std::cout << iter << " " << m.slope << " " << m.intercept << " " << m.error << std::endl;
        best = minModel(m, best);
    }
    std::cout << "error: " << best.error << std::endl;

    if (best.error == std::numeric_limits<double>::infinity())
    {
        std::cerr << "RANSAC failed" << std::endl;
    }
    return best;
}

std::vector<double2> readCSV(const char *path)
{
    std::ifstream file(path);
    std::vector<double2> data;

    std::string line;
    while (getline(file, line))
    {
        double2 p;
        sscanf(line.c_str(), "%lf,%lf", &p.x, &p.y);
        data.push_back(p);
    }
    file.close();

    return data;
}

int main(int argc, char const *argv[])
{
    // srand(time(NULL));
    srand(420);
    std::vector<double2> data = readCSV("points.csv");
    int dataSize = data.size();

    clock_t t0 = clock();
    const int numIters = 20;
    float trainRatio = .3f;
    float wellRatio = .1f;
    double errorThresh = .3;
    struct LinearModel m = ransac(&data[0], dataSize, numIters, errorThresh, (int)(trainRatio * dataSize), (int)(wellRatio * dataSize));
    clock_t t1 = clock();
    double elapsed_secs = double(t1 - t0) / CLOCKS_PER_SEC;

    std::cout << "Best Model (slope, intercept): " << m.slope << " " << m.intercept << std::endl;
    std::cout << "Time taken: " << elapsed_secs << "s" << std::endl;

    // write the results to a file for visualisation
    FILE *f = fopen("results.txt", "w");
    if (f == NULL)
    {
        printf("Error opening file!\n");
        exit(1);
    }
    fprintf(f, "%f %f %f\n", m.slope, m.intercept, m.error);
    fclose(f);
    return 0;
}
