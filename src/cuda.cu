#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <iostream>
#include <fstream>
#include <cmath>
#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <thrust/functional.h>

#include "index.h"

struct dist_functor {
    double slope = 0;
    double intercept = 0;
    dist_functor(const double _slope, const double _intercept)
    {
        slope = _slope;
        intercept = _intercept;
    }

    __host__ __device__
    double operator()(const double2 &p) const
    {
        double dist = std::abs(slope * p.x - p.y + intercept) / std::sqrt(slope * slope + 1);
        return dist * dist;
    }
};

// TODO: use thrust for reduction

struct LinearModel
{
    double slope;
    double intercept;
    double error;
};

struct LinearModel minModel(struct LinearModel a, struct LinearModel b)
{
    return a.error < b.error ? a : b;
}

double distance(struct LinearModel model, double2 p)
{
    return std::abs(model.slope * p.x - p.y + model.intercept) / std::sqrt(model.slope * model.slope + 1);
}

void shuffle(int *arr, int size)
{
    // Fisher Yates Shuffle
    // https://en.wikipedia.org/wiki/Fisher%E2%80%93Yates_shuffle
    for (int i = size - 1; i > 0; i--) 
    { 
        int j = rand() % (i + 1); 
        int tmp = arr[i];
        arr[i] = arr[j];
        arr[j] = tmp;
    }
}

int * buildIndices(int dataSize, int maxIter)
{
    int *indices = (int*)malloc(maxIter * dataSize * sizeof(int));
    for (int iter = 0; iter < maxIter; iter++)
    {
        for (int idx = 0; idx < dataSize; idx++)
        {
            indices[iter * dataSize + idx] = idx;
        }
        shuffle(&indices[iter * dataSize], dataSize);
    }
    return indices;
}

__device__ void triangToTuple(int t, int &x, int &y)
{
    y = (int)((1 + (int)sqrtf(1 + 8 * t)) / 2);
    x = t - (y * (y - 1)) / 2;
}

__global__ void buildModel(double *data, int *indices, int numCombinations, struct LinearModel *models)
{
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    int i, k;
    triangToTuple(t, i, k);

    double *pI = &data[2 * indices[i]];
    double *pK = &data[2 * indices[k]];

    struct LinearModel m;
    m.slope = (pK[1] - pI[1]) / (pK[0] - pI[0]);
    m.intercept = pI[1] - m.slope * pI[0];

    models[t] = m;
}

// __global__ void distances(struct LinearModel model, double *data, int *indices, int trainSize)
// {
//     double2 p;
//     p.x = data[2 * indices[d]];
//     p.y = data[2 * indices[d] + 1];
//     // double dist = distance(candidates[t], p);
//     double dist = std::abs(candidates[t].slope * p.x - p.y + candidates[t].intercept) / std::sqrt(candidates[t].slope * candidates[t].slope + 1);
// }

__global__ void modelDistance(struct LinearModel *candidates, double *data, int *indices, int trainSize)
{
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    double sum = 0;
    // reduce            double dist = std::abs(m.slope * p.x - p.y + m.intercept) / std::sqrt(m.slope * m.slope + 1);

    for (int d = 0; d < trainSize; d++)
    {
        double2 p;
        p.x = data[2 * indices[d]];
        p.y = data[2 * indices[d] + 1];
        // double dist = distance(candidates[t], p);
        double dist = std::abs(candidates[t].slope * p.x - p.y + candidates[t].intercept) / std::sqrt(candidates[t].slope * candidates[t].slope + 1);
        sum += dist * dist;
    }
    // mean squarred error
    candidates[t].error = sum / trainSize;
}

int checkGood(struct LinearModel m, double *data, int dataSize, int *indices, int trainSize, double thresh, int *good)
{
    // reduce
    int goodSize = 0;
    for (int d = trainSize; d < dataSize; d++)
    {
        double2 p;
        p.x = data[2 * indices[d]];
        p.y = data[2 * indices[d] + 1];
        double dist = distance(m, p);
        if (dist < thresh)
        {
            good[goodSize] = d;
            goodSize++;
        }
    }

    return goodSize;
}

struct LinearModel findBest(int trainSize, struct LinearModel *candidates, int candidatesSize, double *data, int *indices)
{
    struct LinearModel bestModel;
    bestModel.error = std::numeric_limits<double>::infinity();

    // modelDistance<<<,>>>(candidates, data, indices, trainSize);

    // reduce
    for (int t = 0; t < candidatesSize; t++)
    {
        struct LinearModel m = candidates[t];
        struct dist_functor dist_op = dist_functor(m.slope, m.intercept);
        thrust::plus<double> add_op;
        double2 dataPoints[trainSize];
        for (int i = 0; i < trainSize; i++)
        {
            dataPoints[i].x = data[2 * indices[i]];
            dataPoints[i].y = data[2 * indices[i] + 1];
        }
        double sum = thrust::transform_reduce(dataPoints, dataPoints + trainSize, dist_op, 0.0, add_op);
        // mean squarred error
        m.error = sum / trainSize;
        candidates[t] = m;
    }
    for (int t = 0; t < candidatesSize; t++)
    {
        bestModel = minModel(candidates[t], bestModel);
    }

    // for (int t = 0; t < candidatesSize; t++)
    // {
    //     struct LinearModel m = candidates[t];
    //     double sum = 0;
    //     // reduce
    //     for (int d = 0; d < trainSize; d++)
    //     {
    //         double2 p;
    //         p.x = data[2 * indices[d]];
    //         p.y = data[2 * indices[d] + 1];
    //         double dist = distance(m, p);
    //         sum += dist * dist;
    //     }
    //     // mean squarred error
    //     m.error = sum / trainSize;

    //     bestModel = minModel(m, bestModel);
    // }
    return bestModel;
}

struct LinearModel singleIter(int iter, int *indices, double *data, int dataSize, double thresh, int trainSize, int wellCount)
{
    int *d_indices;
    hipMalloc(&d_indices, dataSize * sizeof(int));
    hipMemcpy(d_indices, indices, dataSize * sizeof(int), hipMemcpyHostToDevice);

    double *d_data;
    hipMalloc(&d_data, 2 * dataSize * sizeof(double));
    hipMemcpy(d_data, data, 2 * dataSize * sizeof(double), hipMemcpyHostToDevice);

    // the number of unique combinations of all data points
    int numCombinations = triangMax(trainSize);

    int threadsPerBlock = 32;
    int numBlocks = numCombinations / threadsPerBlock;
    // ignore some combinations OR fill them up with duplicates
    numCombinations = numBlocks * threadsPerBlock;

    // produce every possible model
    int candidateSize = numCombinations * sizeof(struct LinearModel);
    struct LinearModel *d_candidateModels;
    hipMalloc(&d_candidateModels, candidateSize);
    
    buildModel<<<numBlocks, threadsPerBlock>>>(d_data, d_indices, numCombinations, d_candidateModels);
    struct LinearModel *candidateModels = (struct LinearModel*)malloc(candidateSize);
    hipMemcpy(candidateModels, d_candidateModels, candidateSize, hipMemcpyDeviceToHost);
    hipFree(d_candidateModels);

    struct LinearModel bestModel = findBest(trainSize, candidateModels, numCombinations, data, indices);

    free(candidateModels);

    // evaluate the models
    int numGood;
    int tmpInlierIndices[dataSize - trainSize];
    numGood = checkGood(bestModel, data, dataSize, indices, trainSize, thresh, tmpInlierIndices);
    if (numGood < wellCount)
    {
        struct LinearModel fail;
        fail.slope = 0;
        fail.intercept = 0;
        fail.error = std::numeric_limits<double>::infinity();
        return fail;
    }
    else
    {
        // reorder (put the new inliers first)
        for (int g = 0; g < numGood; g++)
        {
            int tmp = indices[trainSize + g];
            indices[trainSize + g] = indices[tmpInlierIndices[g]];
            indices[tmpInlierIndices[g]] = tmp;
        }
    }

    int numTrainAndGood = trainSize + numGood;
    int numInlierComb = triangMax(numTrainAndGood);

    // fit again using the new inlier indices
    numBlocks = numInlierComb / threadsPerBlock;
    numInlierComb = numBlocks * threadsPerBlock;
    struct LinearModel *inlierModels = (struct LinearModel*)malloc(numInlierComb * sizeof(struct LinearModel));
    struct LinearModel *d_inlierModels;
    hipMalloc(&d_inlierModels, numInlierComb * sizeof(inlierModels[0]));

    buildModel<<<numBlocks, threadsPerBlock>>>(d_data, d_indices, numInlierComb, d_inlierModels);

    hipMemcpy(inlierModels, d_inlierModels, numInlierComb * sizeof(inlierModels[0]), hipMemcpyDeviceToHost);
    hipFree(d_inlierModels);

    bestModel = findBest(numTrainAndGood, inlierModels, numInlierComb, data, indices);

    free(inlierModels);

    hipFree(d_data);
    hipFree(d_indices);

    return bestModel;
}

struct LinearModel ransac(double *data, int dataSize, int maxIter, double thresh, int trainSize, int wellCount)
{
    int *indices = buildIndices(dataSize, maxIter);

    struct LinearModel best;
    best.error = std::numeric_limits<double>::infinity();
    for (int iter = 0; iter < maxIter; iter++)
    {
        struct LinearModel m = singleIter(iter, &indices[iter * dataSize], data, dataSize, thresh, trainSize, wellCount);
        std::cout << iter << " " << m.slope << " " << m.intercept << " " << m.error << std::endl;
        best = minModel(m, best);
    }
    std::cout << "error: " << best.error << std::endl;

    if (best.error == std::numeric_limits<double>::infinity())
    {
        std::cerr << "RANSAC failed" << std::endl;
    }
    return best;
}

std::vector<double> readCSV(const char *path)
{
    std::ifstream file(path);
    std::vector<double> data;

    std::string line;
    while (getline(file, line))
    {
        double x, y;
        sscanf(line.c_str(), "%lf,%lf", &x, &y);
        data.push_back(x);
        data.push_back(y);
    }
    file.close();

    return data;
}

int main(int argc, char const *argv[])
{
    // srand(time(NULL));
    srand(420);
    std::vector<double> data = readCSV("points.csv");
    int dataSize = data.size() / 2;

    clock_t t0 = clock();
    const int numIters = 20;
    float trainRatio = .3f;
    float wellRatio = .1f;
    double errorThresh = .3;
    struct LinearModel m = ransac(&data[0], dataSize, numIters, errorThresh, (int)(trainRatio * dataSize), (int)(wellRatio * dataSize));
    clock_t t1 = clock();
    double elapsed_secs = double(t1 - t0) / CLOCKS_PER_SEC;

    std::cout << "Best Model (slope, intercept): " << m.slope << " " << m.intercept << std::endl;
    std::cout << "Time taken: " << elapsed_secs << "s" << std::endl;

    // write the results to a file for visualisation
    FILE *f = fopen("results.txt", "w");
    if (f == NULL)
    {
        printf("Error opening file!\n");
        exit(1);
    }
    fprintf(f, "%f %f %f\n", m.slope, m.intercept, m.error);
    fclose(f);
    return 0;
}
