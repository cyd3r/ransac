#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <iostream>
#include <fstream>
#include <cmath>
#include <thrust/transform_reduce.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <thrust/functional.h>

#include "index.h"

void printModel(double slope, double intercept, double error)
{
    std::cout << "slope " << slope << " intercept " << intercept << " " << error << std::endl;
}

double3 getInfModel()
{
    double3 m;
    m.z = std::numeric_limits<double>::infinity();
    return m;
}

struct dist_functor {
    double slope = 0;
    double intercept = 0;
    dist_functor(const double _slope, const double _intercept)
    {
        slope = _slope;
        intercept = _intercept;
    }

    __host__ __device__
    double operator()(const double2 &p) const
    {
        double dist = std::abs(slope * p.x - p.y + intercept) / std::sqrt(slope * slope + 1);
        return dist * dist;
    }
};

struct min_model
{
    __host__ __device__
    double3 operator()(const double3 &a, const double3 &b) const
    {
        return a.z < b.z ? a : b;
    }
};

struct LinearModel
{
    double slope;
    double intercept;
    double error;
};

struct LinearModel minModel(struct LinearModel a, struct LinearModel b)
{
    return a.error < b.error ? a : b;
}

double distance(struct LinearModel model, double2 p)
{
    return std::abs(model.slope * p.x - p.y + model.intercept) / std::sqrt(model.slope * model.slope + 1);
}

template<class T>
void shuffle(T *arr, int size)
{
    // Fisher Yates Shuffle
    // https://en.wikipedia.org/wiki/Fisher%E2%80%93Yates_shuffle
    for (int i = size - 1; i > 0; i--) 
    { 
        int j = rand() % (i + 1); 
        T tmp = arr[i];
        arr[i] = arr[j];
        arr[j] = tmp;
    }
}

__device__ void triangToTuple(int t, int &x, int &y)
{
    y = (int)((1 + (int)sqrtf(1 + 8 * t)) / 2);
    x = t - (y * (y - 1)) / 2;
}

__global__ void buildModel(double2 *data, int numCombinations, struct LinearModel *models)
{
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    int i, k;
    triangToTuple(t, i, k);

    double2 pI = data[i];
    double2 pK = data[k];

    struct LinearModel m;
    m.slope = (pK.y - pI.y) / (pK.x - pI.x);
    m.intercept = pI.y - m.slope * pI.x;

    models[t] = m;
}

int checkGood(struct LinearModel m, double2 *data, int dataSize, int trainSize, double thresh, int *good)
{
    // reduce
    int goodSize = 0;
    for (int d = trainSize; d < dataSize; d++)
    {
        double2 p = data[d];
        double dist = distance(m, p);
        if (dist < thresh)
        {
            good[goodSize] = d;
            goodSize++;
        }
    }

    return goodSize;
}

struct LinearModel findBest(int trainSize, struct LinearModel *candidates, int candidatesSize, double2 *data, double2 *d_data)
{
    double3 cs[candidatesSize];

    // reduce
    for (int t = 0; t < candidatesSize; t++)
    {
        struct LinearModel m = candidates[t];
        struct dist_functor dist_op = dist_functor(m.slope, m.intercept);
        double sum = thrust::transform_reduce(data, data + trainSize, dist_op, 0.0, thrust::plus<double>());
        // double sum = thrust::transform_reduce(thrust::device, d_data, d_data + trainSize, dist_op, 0.0, thrust::plus<double>());
        // mean squarred error
        cs[t].x = m.slope;
        cs[t].y = m.intercept;
        cs[t].z = sum / trainSize;
    }

    struct min_model min_op;
    double3 bestM = thrust::reduce(cs, cs + candidatesSize, getInfModel(), min_op);
    struct LinearModel m;
    m.slope = bestM.x;
    m.intercept = bestM.y;
    m.error = bestM.z;
    return m;
}

struct LinearModel singleIter(int iter, double2 *rawData, int dataSize, double thresh, int wellCount)
{
    // create a copy of the data (maybe not needed in the future)
    double2 data[dataSize];
    for (int i = 0; i < dataSize; i++)
    {
        data[i] = rawData[i];
    }
    shuffle(data, dataSize);
    double2 *d_data;
    hipMalloc(&d_data, dataSize * sizeof(double2));
    hipMemcpy(d_data, data, dataSize * sizeof(double2), hipMemcpyHostToDevice);

    double2 pI = data[0];
    double2 pK = data[1];

    struct LinearModel bestModel;
    bestModel.slope = (pK.y - pI.y) / (pK.x - pI.x);
    bestModel.intercept = pI.y - bestModel.slope * pI.x;


    // evaluate the models
    int numGood;
    int tmpInlierIndices[dataSize - 2];
    numGood = checkGood(bestModel, data, dataSize, 2, thresh, tmpInlierIndices);
    if (numGood < wellCount)
    {
        std::cout << "num good: " << numGood << "<" << wellCount << std::endl;
        struct LinearModel fail;
        fail.slope = std::numeric_limits<double>::quiet_NaN();
        fail.intercept = std::numeric_limits<double>::quiet_NaN();
        fail.error = std::numeric_limits<double>::infinity();
        return fail;
    }
    else
    {
        // reorder (put the new inliers first)
        for (int g = 0; g < numGood; g++)
        {
            double2 tmp = data[2 + g];
            data[2 + g] = data[tmpInlierIndices[g]];
            data[tmpInlierIndices[g]] = tmp;
        }
    }

    bestModel.error = numGood;

    int threadsPerBlock = 256;
    int numTrainAndGood = 2 + numGood;
    int numInlierComb = triangMax(numTrainAndGood);

    // fit again using the new inlier indices
    int numBlocks = numInlierComb / threadsPerBlock;
    numInlierComb = numBlocks * threadsPerBlock;
    struct LinearModel *inlierModels = (struct LinearModel*)malloc(numInlierComb * sizeof(struct LinearModel));
    struct LinearModel *d_inlierModels;
    hipMalloc(&d_inlierModels, numInlierComb * sizeof(inlierModels[0]));

    buildModel<<<numBlocks, threadsPerBlock>>>(d_data, numInlierComb, d_inlierModels);

    hipMemcpy(inlierModels, d_inlierModels, numInlierComb * sizeof(inlierModels[0]), hipMemcpyDeviceToHost);
    hipFree(d_inlierModels);

    bestModel = findBest(numTrainAndGood, inlierModels, numInlierComb, data, d_data);

    free(inlierModels);

    hipFree(d_data);

    return bestModel;
}

struct LinearModel ransac(double2 *data, int dataSize, int maxIter, double thresh, int wellCount)
{
    struct LinearModel best;
    best.error = std::numeric_limits<double>::infinity();
    for (int iter = 0; iter < maxIter; iter++)
    {
        struct LinearModel m = singleIter(iter, data, dataSize, thresh, wellCount);
        std::cout << iter << " " << m.slope << " " << m.intercept << " " << m.error << std::endl;
        best = minModel(m, best);
    }
    std::cout << "error: " << best.error << std::endl;

    if (best.error == std::numeric_limits<double>::infinity())
    {
        std::cerr << "RANSAC failed" << std::endl;
    }
    return best;
}

std::vector<double2> readCSV(const char *path)
{
    std::ifstream file(path);
    std::vector<double2> data;

    std::string line;
    while (getline(file, line))
    {
        double2 p;
        sscanf(line.c_str(), "%lf,%lf", &p.x, &p.y);
        data.push_back(p);
    }
    file.close();

    return data;
}

int main(int argc, char const *argv[])
{
    // srand(time(NULL));
    srand(420);
    std::vector<double2> data = readCSV("points.csv");
    int dataSize = data.size();

    clock_t t0 = clock();
    const int numIters = 7;
    float wellRatio = .05f;
    double errorThresh = .3;
    struct LinearModel m = ransac(&data[0], dataSize, numIters, errorThresh, (int)(wellRatio * dataSize));
    clock_t t1 = clock();
    double elapsed_secs = double(t1 - t0) / CLOCKS_PER_SEC;

    std::cout << "Best Model (slope, intercept): " << m.slope << " " << m.intercept << std::endl;
    std::cout << "Time taken: " << elapsed_secs << "s" << std::endl;

    // write the results to a file for visualisation
    FILE *f = fopen("results.txt", "w");
    if (f == NULL)
    {
        printf("Error opening file!\n");
        exit(1);
    }
    fprintf(f, "%f %f %f\n", m.slope, m.intercept, m.error);
    fclose(f);
    return 0;
}
